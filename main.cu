///////////////////////////////////////////////////////////////////////////
/// PROGRAMACIÓN EN CUDA C/C++
/// Práctica:	BASICO 4 : Arrays Multidimensionales
/// Autor:		Gustavo Gutierrez Martin
/// Fecha:		Octubre 2022
///////////////////////////////////////////////////////////////////////////

/// dependencias ///
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include "hip/hip_runtime.h"

/// constantes ///
#define MB (1<<20) /// MiB = 2^20
#define ROWS 6
#define COLUMNS 21

/// muestra por consola que no se ha encontrado un dispositivo CUDA
int getErrorDevice();
/// muestra los datos de los dispositivos CUDA encontrados
int getDataDevice(int deviceCount);
/// numero de CUDA cores
int getCudaCores(hipDeviceProp_t deviceProperties);
/// muestra por pantalla las propiedades del dispositivo CUDA
int getDeviceProperties(int deviceId, int cudaCores, hipDeviceProp_t cudaProperties);
/// inicializa el array del host
int loadHostData(int *hst_vector1, int *hst_vector2);
/// transferimos los datos del host al device
int dataTransferToDevice(int *hst_vector1, int *dev_vector1);
/// realiza la suma de los arrays en el device
__global__ void transfer(int *dev_vector1, int *dev_vector2);
/// transfiere los datos del device al host
int dataTransferToHost(int *hst_vector2, int *dev_vector2);
/// muestra por pantalla los datos del host
int printData(int *hst_vector1, int *hst_vector2);
/// función que muestra por pantalla la salida del programa
int getAppOutput();

int main() {
    int deviceCount;
    dim3 blocks(1);
    dim3 threads(ROWS, COLUMNS);
    int *hst_vector1, *dev_vector1;
    int *hst_vector2, *dev_vector2;

    /// buscando dispositivos
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        /// mostramos el error si no se encuentra un dispositivo
        return getErrorDevice();
    } else {
        /// mostramos los datos de los dispositivos CUDA encontrados
        getDataDevice(deviceCount);
    }
    /// reserva del espacio de memoria en el host
    hst_vector1 = (int*)malloc( ROWS * COLUMNS * sizeof(int));
    hst_vector2 = (int*)malloc( ROWS * COLUMNS * sizeof(int));
    /// reserva del espacio de memoria en el device
    hipMalloc( (void**)&dev_vector1, ROWS * COLUMNS * sizeof(int) );
    hipMalloc( (void**)&dev_vector2, ROWS * COLUMNS * sizeof(int) );
    /// cargamos los datos del host
    loadHostData(hst_vector1, hst_vector2);
    /// transferimos los datos del host al device
    dataTransferToDevice(hst_vector1, dev_vector1);
    /// mostramos los datos con los que llamamos al device
    printf("Lanzamiento de: %d bloque y %d hilos \n", 1, threads.x * threads.y);
    printf("> Eje X: %d \n", threads.x);
    printf("> Eje Y: %d \n", threads.y);
    printf("***************************************************\n");
    /// sumamos los items
    transfer<<< blocks, threads >>>(dev_vector1, dev_vector2);
    /// transferimos los datos del device al host
    dataTransferToHost(hst_vector2,dev_vector2);
    /// muestra por pantalla los datos del host
    printData(hst_vector1,hst_vector2);
    /// función que muestra por pantalla la salida del programa
    getAppOutput();
    /// liberamos los recursos del device
    hipFree(dev_vector1);
    hipFree(dev_vector2);
    return 0;
}

int getErrorDevice() {
    printf("¡No se ha encontrado un dispositivo CUDA!\n");
    printf("<pulsa [INTRO] para finalizar>");
    getchar();
    return 1;
}

int getDataDevice(int deviceCount) {
    printf("Se han encontrado %d dispositivos CUDA:\n", deviceCount);
    for (int deviceID = 0; deviceID < deviceCount; deviceID++) {
        ///obtenemos las propiedades del dispositivo CUDA
        hipDeviceProp_t deviceProp{};
        hipGetDeviceProperties(&deviceProp, deviceID);
        getDeviceProperties(deviceID, getCudaCores(deviceProp), deviceProp);
    }
    return 0;
}

int getCudaCores(hipDeviceProp_t deviceProperties) {
    int cudaCores = 0;
    int major = deviceProperties.major;
    if (major == 1) {
        /// TESLA
        cudaCores = 8;
    } else if (major == 2) {
        /// FERMI
        if (deviceProperties.minor == 0) {
            cudaCores = 32;
        } else {
            cudaCores = 48;
        }
    } else if (major == 3) {
        /// KEPLER
        cudaCores = 192;
    } else if (major == 5) {
        /// MAXWELL
        cudaCores = 128;
    } else if (major == 6 || major == 7 || major == 8) {
        /// PASCAL, VOLTA (7.0), TURING (7.5), AMPERE
        cudaCores = 64;
    } else {
        /// ARQUITECTURA DESCONOCIDA
        cudaCores = 0;
        printf("¡Dispositivo desconocido!\n");
    }
    return cudaCores;
}

int getDeviceProperties(int deviceId, int cudaCores, hipDeviceProp_t cudaProperties) {
    int SM = cudaProperties.multiProcessorCount;
    printf("***************************************************\n");
    printf("DEVICE %d: %s\n", deviceId, cudaProperties.name);
    printf("***************************************************\n");
    printf("- Capacidad de Computo            \t: %d.%d\n", cudaProperties.major, cudaProperties.minor);
    printf("- No. de MultiProcesadores        \t: %d \n", SM);
    printf("- No. de CUDA Cores (%dx%d)       \t: %d \n", cudaCores, SM, cudaCores * SM);
    printf("- Memoria Global (total)          \t: %zu MiB\n", cudaProperties.totalGlobalMem / MB);
    printf("- No. maximo de Hilos (por bloque)\t: %d\n", cudaProperties.maxThreadsPerBlock);
    printf("***************************************************\n");
    return 0;
}

int loadHostData(int *hst_vector1, int *hst_vector2) {
    srand ( (int)time(nullptr) );
    for (int i=0; i < ROWS * COLUMNS; i++)  {
        /// inicializamos hst_vector1 con numeros aleatorios entre 0 y 1
        hst_vector1[i] = (int) rand() % 10;
    }
    return 0;
}

int dataTransferToDevice(int *hst_vector1, int *dev_vector1) {
    /// transfiere datos de hst_A a dev_A
    hipMemcpy(dev_vector1,hst_vector1, ROWS * COLUMNS * sizeof(int),hipMemcpyHostToDevice);
    return 0;
}

__global__ void transfer(int *dev_vector1, int *dev_vector2) {
    /// identificador del hilo
    unsigned int threadX = threadIdx.y;
    unsigned int threadY = threadIdx.x;
    /// calculamos el ID  hilo
    unsigned int myID = threadY + threadX * blockDim.x;
    /// calculamos la fila donde se encuentra la posicion
    int row = (int) myID / COLUMNS;
    /// calculamos si la posicion
    if ((myID - (row * COLUMNS)) % 2 == 0 ) {
        dev_vector2[myID] = dev_vector1[myID];
    } else {
        dev_vector2[myID] = 0;
    }

}

int dataTransferToHost(int *hst_vector2, int *dev_vector2) {
    /// transfiere datos de dev_vector2 a hst_vector2
    hipMemcpy(hst_vector2, dev_vector2, ROWS * COLUMNS * sizeof(int), hipMemcpyDeviceToHost);
    return 0;
}

int printData(int *hst_vector1, int *hst_vector2) {
    printf("MATRIZ ORIGINAL:\n");
    for (int i = 0; i < ROWS; i++)  {
        for (int j = 0; j < COLUMNS; j++) {
            printf("%d ", hst_vector1[j + i * COLUMNS]);
        }
        printf("\n");
    }
    printf("\n");
    printf("MATRIZ FINAL:\n");
    for (int i = 0; i < ROWS; i++)  {
        for (int j = 0; j < COLUMNS; j++) {
            printf("%d ", hst_vector2[j + i * COLUMNS]);
        }
        printf("\n");
    }
    printf("\n");
    return 0;
}

int getAppOutput() {
    /// salida del programa
    time_t fecha;
    time(&fecha);
    printf("***************************************************\n");
    printf("Programa ejecutado el: %s", ctime(&fecha));
    printf("***************************************************\n");
    /// capturamos un INTRO para que no se cierre la consola de MSVS
    printf("<pulsa [INTRO] para finalizar>");
    getchar();
    return 0;
}
